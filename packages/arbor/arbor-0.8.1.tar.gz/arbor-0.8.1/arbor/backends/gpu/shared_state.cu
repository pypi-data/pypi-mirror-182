#include "hip/hip_runtime.h"
// GPU kernels and wrappers for shared state methods.

#include <cstdint>

#include <backends/event.hpp>
#include <backends/multi_event_stream_state.hpp>

#include <arbor/gpu/gpu_api.hpp>
#include <arbor/gpu/gpu_common.hpp>

namespace arb {
namespace gpu {

namespace kernel {

template <typename T>
__global__ void update_time_to_impl(unsigned n,
                                    T* __restrict__ const time_to,
                                    const T* __restrict__ const time,
                                    T dt,
                                    T tmax) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        auto t = time[i]+dt;
        time_to[i] = t<tmax? t: tmax;
    }
}

// Vector/scalar addition: x[i] += v ∀i
template <typename T>
__global__ void add_scalar(unsigned n,
                           T* __restrict__ const x,
                           arb_value_type v) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        x[i] += v;
    }
}

template <typename T, typename I>
__global__ void set_dt_impl(      T* __restrict__ dt_intdom,
                            const T* __restrict__ time_to,
                            const T* __restrict__ time,
                            const unsigned ncomp,
                                  T* __restrict__ dt_comp,
                            const I* __restrict__ cv_to_intdom) {
    auto idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < ncomp) {
        const auto ind = cv_to_intdom[idx];
        const auto dt = time_to[ind] - time[ind];
        dt_intdom[ind] = dt;
        dt_comp[idx] = dt;
    }
}

__global__ void take_samples_impl(
    multi_event_stream_state<raw_probe_info> s,
    const arb_value_type* __restrict__ const time,
    arb_value_type* __restrict__ const sample_time,
    arb_value_type* __restrict__ const sample_value)
{
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<s.n) {
        auto begin = s.ev_data+s.begin_offset[i];
        auto end = s.ev_data+s.end_offset[i];
        for (auto p = begin; p!=end; ++p) {
            sample_time[p->offset] = time[i];
            sample_value[p->offset] = p->handle? *p->handle: 0;
        }
    }
}

} // namespace kernel

using impl::block_count;

void add_scalar(std::size_t n, arb_value_type* data, arb_value_type v) {
    if (!n) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(n, block_dim);
    kernel::add_scalar<<<nblock, block_dim>>>(n, data, v);
}

void update_time_to_impl(
    std::size_t n, arb_value_type* time_to, const arb_value_type* time,
    arb_value_type dt, arb_value_type tmax)
{
    if (!n) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(n, block_dim);
    kernel::update_time_to_impl<<<nblock, block_dim>>>(n, time_to, time, dt, tmax);
}

void set_dt_impl(
    arb_size_type nintdom, arb_size_type ncomp, arb_value_type* dt_intdom, arb_value_type* dt_comp,
    const arb_value_type* time_to, const arb_value_type* time, const arb_index_type* cv_to_intdom)
{
    if (!nintdom || !ncomp) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(ncomp, block_dim);
    kernel::set_dt_impl<<<nblock, block_dim>>>(dt_intdom, time_to, time, ncomp, dt_comp, cv_to_intdom);
}

void take_samples_impl(
    const multi_event_stream_state<raw_probe_info>& s,
    const arb_value_type* time, arb_value_type* sample_time, arb_value_type* sample_value)
{
    if (!s.n_streams()) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(s.n_streams(), block_dim);
    kernel::take_samples_impl<<<nblock, block_dim>>>(s, time, sample_time, sample_value);
}

} // namespace gpu
} // namespace arb
